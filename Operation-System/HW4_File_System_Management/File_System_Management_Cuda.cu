
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <inttypes.h>
#include<string.h>
#include <sys/time.h>
#include <time.h>


///#define STORAGE_SIZE 1085440
#define LOG_ENABLE 0
#define MAX_FILE_SIZE 1048576
#define DATAFILE "./data.bin"
#define OUTFILE "./snapshot.bin"
#define KBytes 1024
#define MAX_NUM_OF_FILE 1024
#define fileNameLength 20
#define FREE_BIT 0
#define USED_BIT 1

#define G_READ 0
#define G_WRITE 1
#define RM 2
#define LS_S 3
#define LS_D 4

#define ERROR_LIST_FILE       904000000
#define ERROR_FCB_FULL        903000000
#define ERROR_FILE_BLOCK_FULL 902000000
#define ERROR_FILE_NOT_EXIST  901000000
#define U32_NOT_NUMBER  900000000 ///%u
#define U16_NOT_NUMBER  65000 /// 0~ 65535(六萬五千) (%hu)

typedef unsigned char uchar;  ///1byte
typedef uint32_t u32;         ///4bytes, 32bits
typedef uint16_t u16;

///uchar *volume;

__device__ __managed__ uchar bitVector[MAX_NUM_OF_FILE];  /// 1K =  1* 1024
__device__ __managed__ uchar *file_content;  /// 1024K = 1024 * 1024
__device__ __managed__ u16 fileCreatedTimeCount =0;  /// 16 bytes
__device__ __managed__ u16 fileModifiedTimeCount =0; ///  16 bytes

#pragma pack(push) ///把原本的對齊設定push進stack
#pragma pack(1)
typedef struct FCB{         /// total=  28 byte
   char file_name[fileNameLength];  /// 20 byte
   u16 fileSize;              ///        2 byte
   u16 fileCreatedTime;        ///       2 byte
   u16 fileModifiedTime;  ///            2 byte
   u16 file_content_index;   ///         2 byte
};
#pragma pack(pop)  ///把原本的對齊設定pop出來

__device__ __managed__ struct FCB *fcbs;   /// 28K =28 *1024

__device__  void swapFCB(int iFcbIndex,int jFcbIndex);

__device__  void initCharVaule(char *names,char value, int size ){
   int i=0;
   for (i=0;i<=size;i++){
       names[i]=value;
   }
}

 __device__ void init_volume(){

  int i=0;
  for (i=0;i<MAX_NUM_OF_FILE;i++ ){
      ///printf("i=%d ",i);
      bitVector[i]=FREE_BIT;   /// 0 != '0'
      /// if (bitVector[i] == 0) {}
      ///printf("bitVector[%d]=%d; ",i,bitVector[i]); ///%d or %u


      initCharVaule(fcbs[i].file_name,'\0',fileNameLength );   /// 0 == '\0'  but != '0'
      fcbs[i].fileSize=U16_NOT_NUMBER;
      fcbs[i].fileCreatedTime=U16_NOT_NUMBER;
      fcbs[i].fileModifiedTime=U16_NOT_NUMBER;
      fcbs[i].file_content_index=U16_NOT_NUMBER;
  }

  ///printf("fileSize[%d]=%d; \n", (KBytes-1) ,fileSize[(KBytes-1)] ); ///%d or %u
  ///printf("fileCreatedTime[%d]=%d; \n", (KBytes-1) ,fileCreatedTime[(KBytes-1)] );
  ///printf("fileModifiedTimeSize[%d]=%d; \n", (KBytes-1) ,fileModifiedTimeSize[(KBytes-1)] );



  for (i=0;i<MAX_FILE_SIZE;i++ ){
        file_content[i]='\0';
  }

  ///printf("file_content[%d]=%d; \n",(MAX_FILE_SIZE-1),file_content[(MAX_FILE_SIZE-1)]); ///%d or %u


}


__device__  int stringLength(const char *symbols){
    int i=0;
    for (i=0;symbols[i] !='\0';i++ ){
    }
    return i;

}


__device__  void gpuStrncpy(const char *source,  char *destination, int size  ){

    int i=0;
    char sChar = *source;

     while( sChar !='\0' && i<size  ){
        *destination = sChar;
        destination++;
        source++;
        sChar = *source;
        i++;
    }

    /**
         /// Key !! when convert c code to cuda code.
         /// 1. in cuda cannot use pointer array!!! like  *(source+i)
         /// 2. *destination will be assigned value like destination = '\0',
         ///     so *destination   cannot declare const!!!

    while( *(source+i) !='\0' && i<size  ){
        *destination = *(source+i);
        destination++;
        i++;
    }**/

    ///if ( *(source+i) == '\0'){
       *destination = '\0';
    ///}

}

__device__  int gpuStrcmp( const char *str1, const char *str2){

   ///printf("str1=%s, str2=%s, *(str2+1)=%c, str2[1]=%c, *str2=%c,\n,",&str1[0],str2,*(str2+1),str2[1],*str2 );
   ///printf("str1=%s, str2=%s, str2[1]=%c\n,",&str1[0],str2,str2[1]);
   /// str1 == str2 = 0, str1 > str2 =  >0 , str1 <str2 =  <0
   char s1,s2;

   s1 = *str1;
   s2 = *str2;


   ///printf("1. s1=%c,s2=%c,s1= %d,s2=%d \n",s1,s2,s1,s2);
   ///if (s1 == NULL || s2 ==NULL){
   ///   printf("s1 == NULL or s2 == NULL\n");
   ///}



   while (s1 ==s2){
        ///printf("s1=%c,s2=%c,s1= %d,s2=%d \n",s1,s2,s1,s2);
       if (s1 == '\0' || s2 =='\0'){
           break;
       }
       ++str2;
       s2= *str2;
       ++str1;
       s1 = *str1;
   }

   return ((const unsigned char )s1 - (const unsigned char)(s2));

}



__device__  u16 findFreeFCB(){
    int i=0;
    for (i=0;i<MAX_NUM_OF_FILE;i++){
        if (fcbs[i].file_name[0] =='\0'){
            return i;
        }
    }
    return U16_NOT_NUMBER;
}

__device__  u16 findFileByName(const char* fileName) {
   int i=0;

   for (i=0;i<MAX_NUM_OF_FILE;i++){
       if ( fcbs[i].file_name[0] == '\0') {
            #if LOG_ENABLE
             ///printf("file_name=0, fcbs[%d].file_name=%s,\n",i,fcbs[i].file_name);
            #endif
            continue; /// unused file.
       }
       if (gpuStrcmp(fileName, fcbs[i].file_name) ==0 ){
            #if LOG_ENABLE
             printf("find fileName=%s, fcbs[%d].file_name=%s, index=%hu\n",fileName,i,fcbs[i].file_name,fcbs[i].file_content_index);
            #endif
           return i;

       }
   }
   return U16_NOT_NUMBER;
}

__device__  u16 findFreeBitVector(){
    int i=0;
    for (i=0;i<MAX_NUM_OF_FILE;i++){
        if (bitVector[i] == FREE_BIT){
            return i;
        }
    }

    return U16_NOT_NUMBER;
}

__device__  void shiftFileCreatedTimeCountForOverFlow(){

    #if LOG_ENABLE
    fprintf(logFp ,"[shiftFileCreatedTimeCountForOverFlow] fileCreatedTimeCount=%hu,\n",fileCreatedTimeCount );
    fflush(stderr);
    #endif

   int i,j=0;

   for (i=0;i< MAX_NUM_OF_FILE;i++){
         if (fcbs[i].file_name[0] == '\0' ) continue;
         for (j=0;j<MAX_NUM_OF_FILE ;j++){
             if (fcbs[j].file_name[0] == '\0' || i==j ) continue;
             /// from small to large <
             /// from large to small >
             if ( fcbs[i].fileCreatedTime  < fcbs[j].fileCreatedTime ){
                 swapFCB(i,j);
             }
         }
   }

   fileCreatedTimeCount =1;

    for (i=0;i< MAX_NUM_OF_FILE;i++){
         if (fcbs[i].file_name[0] == '\0' ) continue;

          #if LOG_ENABLE
          fprintf(logFp ,"fcbs[%d]=%s, fileModifiedTime=%hu, shift from fileCreatedTime=%hu, to fileCreatedTime=%hu\n",i,fcbs[i].file_name,fcbs[i].fileModifiedTime  ,fcbs[i].fileCreatedTime,fileCreatedTimeCount );
          fflush(stderr);
          #endif

          fcbs[i].fileCreatedTime =fileCreatedTimeCount;
          fileCreatedTimeCount++;
    }
}


__device__  u16 createFile(u16 bitLocation, const char *fileName){

    u16 fcbIndex = findFreeFCB();

    if (fcbIndex == U16_NOT_NUMBER){
        return U16_NOT_NUMBER;
    }

    fileCreatedTimeCount++;

    bitVector[bitLocation] = USED_BIT;

    int file_name_length = stringLength(fileName);
    gpuStrncpy(fileName,fcbs[fcbIndex].file_name, file_name_length);
    fcbs[fcbIndex].fileCreatedTime = fileCreatedTimeCount;
    fcbs[fcbIndex].fileModifiedTime = fileCreatedTimeCount;
    fcbs[fcbIndex].fileSize=0;
    fcbs[fcbIndex].file_content_index= bitLocation;

    #if LOG_ENABLE
    printf("file_name_length=%d, fcbs[%d].file_name=%s, fileCreatedTime=%hu, fileSize=%hu, file_content_index=%hu, \n",file_name_length,fcbIndex,fcbs[fcbIndex].file_name,fcbs[fcbIndex].fileCreatedTime, fcbs[fcbIndex].fileSize , fcbs[fcbIndex].file_content_index );
    #endif

     if (fileCreatedTimeCount >= U16_NOT_NUMBER){
        shiftFileCreatedTimeCountForOverFlow();
    }

    return fcbIndex;
}

__device__  u32 open(const char* fileName, int accessMode){

   u16 fcbIndex = findFileByName(fileName);
   u16 bitIndex = U16_NOT_NUMBER;


   if (fcbIndex ==U16_NOT_NUMBER){
       if (accessMode == G_READ){
            return  ERROR_FILE_NOT_EXIST;
       }else{
          bitIndex = findFreeBitVector();
          if (bitIndex == U16_NOT_NUMBER){
             return ERROR_FILE_BLOCK_FULL;
          }
          fcbIndex = createFile(bitIndex , fileName);
          if (fcbIndex == U16_NOT_NUMBER){
              return ERROR_FCB_FULL;
          }
       }
   }

   return (u32) fcbIndex;

}

__device__ void clearFileContent(u32 fcbIndex){
    u16 fb = fcbIndex;
    int i=0;

    if (fcbs[fb].fileSize ==0) {
       #if LOG_ENABLE
       printf( "##clearFileContent## fileSize==0, fcbIndex=%u, fb=%hu, fileSize=%hu\n",fcbIndex,fb,fcbs[fb].fileSize );
       #endif
       return;
    }

    i = fcbs[fb].file_content_index * KBytes;

    #if LOG_ENABLE
    printf("fcbIndex=%u, fb=%hu, fileSize=%hu, file_content_index=%hu, real location= (file_content_index * KBytes)=%d,  \n",fcbIndex,fb,fcbs[fb].fileSize,fcbs[fb].file_content_index, i );
    #endif

    for (;i<fcbs[fb].fileSize;i++){
        file_content[i]='\0';
    }

}


__device__ void shiftFileModifiedTimeCountForOverFlow(){

    #if LOG_ENABLE
    fprintf(logFp ,"[shiftFileModifiedTimeCountForOverFlow] fileModifiedTimeCount=%hu,\n",fileModifiedTimeCount );
    fflush(stderr);
    #endif

   int i,j=0;

   for (i=0;i< MAX_NUM_OF_FILE;i++){
         if (fcbs[i].file_name[0] == '\0' ) continue;
         for (j=0;j<MAX_NUM_OF_FILE ;j++){
             if (fcbs[j].file_name[0] == '\0' || i==j ) continue;
             /// from small to large <
             /// from large to small >
             if ( fcbs[i].fileModifiedTime  < fcbs[j].fileModifiedTime ){
                 swapFCB(i,j);
             }
         }
   }

   fileModifiedTimeCount =1;

    for (i=0;i< MAX_NUM_OF_FILE;i++){
         if (fcbs[i].file_name[0] == '\0' ) continue;
         #if LOG_ENABLE
         fprintf(logFp ,"fcbs[%d]=%s, fileCreatedTime=%hu, shift from fileModifiedTime=%hu, to fileModifiedTime=%hu\n",i,fcbs[i].file_name,fcbs[i].fileCreatedTime  ,fcbs[i].fileModifiedTime,fileModifiedTimeCount );
         fflush(stderr);
         #endif
         fcbs[i].fileModifiedTime =fileModifiedTimeCount;
         fileModifiedTimeCount++;
    }
}


__device__ u32 write(uchar *input, u32 size, u32 fcbIndex){
    u16 fb = U16_NOT_NUMBER;
    int i =0;
    int fileBlockLocation =0;


    if (fcbIndex >= U32_NOT_NUMBER) {
          # if LOG_ENABLE
        printf("[Error:write] fcbIndex=%u, fb=%hu >= U32_NOT_NUMBER\n",fcbIndex,fb );
         #endif

        return fcbIndex;
    }

    clearFileContent(fcbIndex);

    fileModifiedTimeCount++;

    fb = fcbIndex;

    fileBlockLocation =  fcbs[fb].file_content_index * KBytes;

    if (size > KBytes){
         # if LOG_ENABLE
        printf("[Waring] fcbIndex=%u, fb=%hu, size=%u > 1024,\n",fcbIndex,fb,size );
         #endif
        size = KBytes;
    }

    for (;i<size;i++){
        file_content[fileBlockLocation + i] = input[i];
    }

    fcbs[fb].fileModifiedTime= fileModifiedTimeCount;
    fcbs[fb].fileSize = (u16) size;


    if (fileModifiedTimeCount >= U16_NOT_NUMBER){
        shiftFileModifiedTimeCountForOverFlow();
    }


    return size;
}


__device__ u32 read(uchar *output, u32 size, u32 fcbIndex){
    u16 fb = fcbIndex;
    int i =0;
    int fileBlockLocation =0;

    if (fcbIndex >= U32_NOT_NUMBER) {
         # if LOG_ENABLE
        printf("[Error:read] fcbIndex=%u, fb=%hu >= U32_NOT_NUMBER\n",fcbIndex,fb );
         #endif
        return fcbIndex;
    }

    fileBlockLocation =  fcbs[fb].file_content_index * KBytes;

    if (size > KBytes){
         # if LOG_ENABLE
        printf("[Waring.1] fcbIndex=%u, fb=%hu, size=%u > 1024,\n",fcbIndex,fb,size );
         #endif
        size = KBytes;
    }

    if (size > fcbs[fb].fileSize ){
         # if LOG_ENABLE
        printf("[Waring.2] fcbIndex=%u, fb=%hu, size=%u > fileSize=%hu,\n",fcbIndex,fb,size,fcbs[fb].fileSize );
         #endif

        size = fcbs[fb].fileSize;
    }

     for (;i<size;i++){
          output[i] = file_content[ (fileBlockLocation + i)];
     }
     return size;

}

__device__  u32 gsys(int mode, const char* fileName ){
    /// do remove file
    u16 fcbIndex = findFileByName(fileName);

    if (fcbIndex == U16_NOT_NUMBER){
         # if LOG_ENABLE
        printf("[Error:gsy(RM)] List file Error. fcbIndex=%hu,\n",fcbIndex );
         #endif

        return ERROR_LIST_FILE;
    }

    clearFileContent(fcbIndex);

    initCharVaule(fcbs[fcbIndex].file_name,'\0',fileNameLength );
    fcbs[fcbIndex].fileSize  = U16_NOT_NUMBER;
    fcbs[fcbIndex].fileCreatedTime=U16_NOT_NUMBER;
    fcbs[fcbIndex].fileModifiedTime=U16_NOT_NUMBER;

    /// free bitVector before free fcbs.file_content_index
    bitVector[fcbs[fcbIndex].file_content_index] = FREE_BIT;


    fcbs[fcbIndex].file_content_index=U16_NOT_NUMBER;


    return 1;
}


__device__  void swapFCB(int iFcbIndex,int jFcbIndex){
    # if LOG_ENABLE
       printf("[swapFCB] iFcbIndex=%d,jFcbIndex=%d,\n",iFcbIndex,jFcbIndex);
      #endif

    struct FCB fcb;
    initCharVaule(fcb.file_name,'\0',fileNameLength );   /// 0 == '\0'  but != '0'
    fcb.fileSize=U16_NOT_NUMBER;
    fcb.fileCreatedTime=U16_NOT_NUMBER;
    fcb.fileModifiedTime=U16_NOT_NUMBER;
    fcb.file_content_index=U16_NOT_NUMBER;

    /// copy j data to temp
    gpuStrncpy(fcbs[jFcbIndex].file_name,fcb.file_name , stringLength(fcbs[jFcbIndex].file_name));
    fcb.fileSize = fcbs[jFcbIndex].fileSize;
    fcb.file_content_index = fcbs[jFcbIndex].file_content_index;
    fcb.fileCreatedTime = fcbs[jFcbIndex].fileCreatedTime;
    fcb.fileModifiedTime = fcbs[jFcbIndex].fileModifiedTime;

     # if LOG_ENABLE
    printf("j=%d puts into temp, filename=%s, filesize=%u, file_content_index=%hu, fileCreatedTime=%hu, fileModifiedTime=%hu, \n",jFcbIndex,fcb.file_name ,fcb.fileSize ,fcb.file_content_index , fcb.fileCreatedTime,  fcb.fileModifiedTime );
     #endif

    /// copy i data to j
    gpuStrncpy(fcbs[iFcbIndex].file_name, fcbs[jFcbIndex].file_name , stringLength(fcbs[iFcbIndex].file_name));
    fcbs[jFcbIndex].fileSize = fcbs[iFcbIndex].fileSize;
    fcbs[jFcbIndex].file_content_index = fcbs[iFcbIndex].file_content_index;
    fcbs[jFcbIndex].fileCreatedTime = fcbs[iFcbIndex].fileCreatedTime;
    fcbs[jFcbIndex].fileModifiedTime = fcbs[iFcbIndex].fileModifiedTime;

     # if LOG_ENABLE
    printf("i=%d put into j=%d, filename=%s, filesize=%u, file_content_index=%hu, fileCreatedTime=%hu, fileModifiedTime=%hu, \n",iFcbIndex,jFcbIndex,fcbs[jFcbIndex].file_name ,fcbs[jFcbIndex].fileSize ,fcbs[jFcbIndex].file_content_index , fcbs[jFcbIndex].fileCreatedTime,  fcbs[jFcbIndex].fileModifiedTime );
     #endif

    /// copy temp data to i
    gpuStrncpy(fcb.file_name, fcbs[iFcbIndex].file_name , stringLength(fcb.file_name));
    fcbs[iFcbIndex].fileSize = fcb.fileSize;
    fcbs[iFcbIndex].file_content_index = fcb.file_content_index;
    fcbs[iFcbIndex].fileCreatedTime = fcb.fileCreatedTime;
    fcbs[iFcbIndex].fileModifiedTime = fcb.fileModifiedTime;

     # if LOG_ENABLE
    printf("put temp to i=%d, filename=%s, filesize=%u, file_content_index=%hu, fileCreatedTime=%hu, fileModifiedTime=%hu, \n",iFcbIndex,fcbs[iFcbIndex].file_name ,fcbs[iFcbIndex].fileSize ,fcbs[iFcbIndex].file_content_index , fcbs[iFcbIndex].fileCreatedTime,  fcbs[iFcbIndex].fileModifiedTime );
      #endif
}


__device__  u32 gsys(int mode){
   int i,j=0;

   if (mode == LS_D){
       # if LOG_ENABLE
       printf("[gsy LS_D]\n");
      #endif
      /// sort by modified time
      for (i=0;i< MAX_NUM_OF_FILE;i++){
         if (fcbs[i].file_name[0] == '\0' ) continue;
         for (j=0;j<MAX_NUM_OF_FILE ;j++){
             if (fcbs[j].file_name[0] == '\0' || i==j ) continue;
             if ( fcbs[i].fileModifiedTime  > fcbs[j].fileModifiedTime ){
                 swapFCB(i,j);
             }
         }
      }

     printf("===sort by modified time===\n");

   }else if (mode == LS_S){
       # if LOG_ENABLE
       printf("[gsy LS_S]\n");
      #endif
      /// sort by size if size is the same, sort by create time
        for (i=0; i< MAX_NUM_OF_FILE; i++){
            if (fcbs[i].file_name[0] == '\0' ) continue;
            for (j=0; j<MAX_NUM_OF_FILE ; j++){
                if (fcbs[j].file_name[0] == '\0' || i==j ) continue;

                if ( fcbs[i].fileSize  > fcbs[j].fileSize ){
                    swapFCB(i,j);
                }else if ( fcbs[i].fileSize == fcbs[j].fileSize){
                     # if LOG_ENABLE
                    printf("i=%d, j=%d, size the same. size=%hu,\n",i,j,fcbs[i].fileSize);
                     #endif
                    if ( fcbs[i].fileCreatedTime  < fcbs[j].fileCreatedTime ){
                        swapFCB(i,j);
                    }
                }
            }
        }

        printf("===sort by file size===\n");
   }

   /// print sorted result
   for (i=0;i<MAX_NUM_OF_FILE;i++){
        if (fcbs[i].file_name[0] == '\0' ) continue;
        if (mode == LS_D){
            printf("%s\n",fcbs[i].file_name );
        }else if (mode == LS_S){
            printf("%s %hu\n",fcbs[i].file_name, fcbs[i].fileSize );
        }
   }

   return 1;

}



__global__  void mykernel(uchar *input,uchar *output){

     init_volume();


     //####kernel start####
     u32 fp =open("t.txt\0",G_WRITE);
     write(input,64, fp);
     fp =open("b.txt\0",G_WRITE);
     write(input+32,32, fp);
     fp = open("t.txt\0",G_WRITE);
     write(input+32,32, fp);
     fp = open("t.txt\0",G_READ);
     read(output, 32,fp);
     gsys(LS_D);
     gsys(LS_S);
     fp = open("b.txt\0",G_WRITE);
     write(input+64,12, fp);
     gsys(LS_S);
     gsys(LS_D);
     gsys(RM,"t.txt\0");
     gsys(LS_S);
     //####kernel end####
}

void load_binaryFile(char *path, uchar *buffer, int size ){

	FILE *fp;
	fp = fopen(path,"rb");
	//fseek (fp , 0 , SEEK_END);
	//int totalSize = ftell (fp);
	//rewind (fp);
	//int dataSize = totalSize / sizeof( uchar);
	size_t readResultSize = fread(&buffer[0], sizeof(uchar), size , fp);
	//printf("[load_binaryFile] input size=%d, dataSize=%d, readResultSize=%d\n",size,dataSize,readResultSize);
	//printf("[load_binaryFile] input size=%d, readResultSize=%d\n",size,readResultSize);
	fclose(fp);

	 # if LOG_ENABLE
    printf("[load_binaryFile] input size=%d, readResultSize=%d\n",size,readResultSize);
    #endif


    /*
    int* returnSize= (int*) malloc ( sizeof(int));
    returnSize[0] = (int)readResultSize;
    return returnSize[0];
    */
}

void write_binaryFile(char *path, uchar *buffer, int size){
    FILE *outfp;
    outfp = fopen(path,"wb");

    //fseek (outfp , 0 , SEEK_SET);
    //int totalSize = ftell (outfp);
	//rewind (outfp);
	//int dataSize = totalSize / sizeof( uchar);
    size_t writeResultSize = fwrite ( &buffer[0], sizeof(uchar), size, outfp);

    //printf("[write_binaryFile] input size=%d,dataSize=%d,writeResultSize=%d\n",size,dataSize,writeResultSize);
    //printf("[write_binaryFile] input size=%d, writeResultSize=%d\n",size,writeResultSize);

     # if LOG_ENABLE
    printf("[write_binaryFile] input size=%d, writeResultSize=%d\n",size,writeResultSize);
    #endif

    fclose(outfp);
}



int mainRun()
{

    hipSetDevice(1);

    hipMallocManaged(&file_content, MAX_FILE_SIZE);
    hipMallocManaged(&fcbs, sizeof(struct FCB)* MAX_NUM_OF_FILE);
    printf("fcb size=%d bytes, total:(fcb*1024)=%d bytes,\n\n",sizeof(struct FCB),sizeof(struct FCB)* MAX_NUM_OF_FILE );


    uchar *input, *output;
    hipMallocManaged(&input, MAX_FILE_SIZE);
    hipMallocManaged(&output, MAX_FILE_SIZE);


    int i=0;
    for(i=0;i<MAX_FILE_SIZE;i++){
        output[i]=0;
    }

    load_binaryFile(DATAFILE,input,MAX_FILE_SIZE);


    mykernel<<<1,1>>>(input,output);
    hipDeviceSynchronize();

    write_binaryFile(OUTFILE,output,MAX_FILE_SIZE);
    hipDeviceReset();

    printf("\nFinish!\n");

    return 0;
}





int main(){
    struct timeval tv, tv2;

    clock_t endTime;
    unsigned long long start_utime, end_utime;


    endTime =clock();
    gettimeofday(&tv, NULL);

    mainRun();

    gettimeofday(&tv2, NULL);
    endTime =clock() - endTime ;

    start_utime = tv.tv_sec * 1000000 + tv.tv_usec;
	end_utime = tv2.tv_sec * 1000000 + tv2.tv_usec;

    printf("Clock=%f sec. ,  Gettimeofday time = %llu.%03llu milisecond;  %llu.%03llu sec \n",((float)endTime) /CLOCKS_PER_SEC, (end_utime - start_utime)/1000, (end_utime - start_utime)%1000, (end_utime - start_utime)/1000000, (end_utime - start_utime)%1000000  );

     return 0;

}

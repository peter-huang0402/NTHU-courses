
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <sys/time.h>
#include <sys/types.h>
#define DEBUG_ENABLE 0
#define ERROR_TRACING 0
#define V 7000
#define INF 1000000
///const int INF = 1000000;
///const int V = 7000;
void input(char *inFileName);
void output(char *outFileName);

void block_APSP(int B);
int iceil(int a, int b);
void cal(int B, int Round, int block_start_x, int block_start_y, int block_width, int block_height);

int n, m;	// Number of vertices, edges
int Dist[V][V];
int* devDist;
int gpuID=0;
///int Dist[7000][7000];
FILE *logFp;
char *in1;
char *out1;
static int totalCUDADevice = 0;
static int totalNode = 1;
static int MPIID=0;
static int currentDev=0;
typedef struct Timer {
    char name[256];
    struct timeval begin;
    struct timeval end;
} Timer;

 Timer *timer_memcpy,*timer_commu,*timer_compute;
 Timer *timer_phase3;

Timer* timer_init(Timer* t,const char* name);
Timer* timer_new(const char* name);
void timer_start(Timer* t);
void timer_end(Timer* t);
void timer_add(Timer* t1, const Timer* t2);
double timer_seconds(const Timer* t);
void timer_print(const Timer* t,FILE* stream);
void timer_delete(Timer* t);


Timer* timer_init(Timer* t,const char* name ) {
	
    /// memset 介紹 及 常見問題
    /// http://baike.baidu.com/view/982208.htm
    /// 一般情況下，清空 struct的方法︰
    /// t. name [0]={'\0'};
    /// t. begin=0;
    /// t. end=0;
    /// 可用 memset 直接清除，方便。
    /// memset (t,0,sizeof(Timer));
	
    if(t) {
        memset (t,0,sizeof(Timer));
        strncpy(t->name,name,256);
    }
    return t;
}

Timer* timer_new(const char* name) {
    Timer* t;
    t = (Timer*)malloc(sizeof(Timer));
    return timer_init(t,name);
}
void timer_start(Timer* t) {
    if(!t) return;
    gettimeofday(&t->begin,0);
}
void timer_end(Timer* t) {
    if(!t) return;
    gettimeofday(&t->end,0);
}
void timer_add(Timer* t1, const Timer* t2) {
    if(!t1 || !t2) return;
    t1->end.tv_sec+=(t2->end.tv_sec-t2->begin.tv_sec);     /// seconds
    t1->end.tv_usec+=(t2->end.tv_usec-t2->begin.tv_usec);  /// microseconds
}

double timer_seconds(const Timer* t) {
    if(!t) return 0;
    return (double)(t->end.tv_sec-t->begin.tv_sec)+(1e-6*(t->end.tv_usec-t->begin.tv_usec));
}
void timer_delete(Timer* t) {
    if(!t) return;
    free(t);
}
void timer_print(const Timer* t, FILE* stream) {
    if(!t) return;
    fprintf(stream,"%s : %f(sec)\n",t->name,timer_seconds(t));
}

static void __debugCUDACall(hipError_t err, const char* expr, const char* file, int line) {
    if(err != hipSuccess) {
        fprintf(stderr,"in File %s Line %d:%s\n",file,line,expr);
        fprintf(stderr,"%s \n",hipGetErrorString(err));
    }
}
#define debugCUDACall(X) __debugCUDACall((X),#X,__FILE__,__LINE__)

char *stringConcat(char *str1, char *str2) {

    int length=strlen(str1)+strlen(str2)+1;

    char *result = (char*)malloc(sizeof(char) * length);

    // 複製第一個字串至新的陣列空間
    strcpy(result, str1);
    // 串接第二個字串至新的陣列空間
    strcat(result, str2);

    return result;
}


void initCUDADevice(int gpuID)
{
    // Task 1: Device Initialization
    hipGetDeviceCount(&totalCUDADevice);
    printf("totalCUDADevice=%d, \n",totalCUDADevice);


    if (totalCUDADevice == 0) {
        printf("No CUDA device found.\n\n");
    } else if (gpuID < totalCUDADevice) {
        printf("set CUDA device=%d, \n",gpuID );
        hipSetDevice(gpuID);
    } else {
        gpuID =0;
        printf("set CUDA device=%d, \n",gpuID );
        hipSetDevice(gpuID);
    }

}

int mainRun(int argc, char* argv[])
{



    if(argc > 4) {
        sscanf(argv[4],"%d",&gpuID);
    }

    initCUDADevice(gpuID);

    if(argc > 5) {
        sscanf(argv[5],"%d",&totalCUDADevice);
        printf("### set totalCUDADevice=%d by argv[5] \n",totalCUDADevice);
        if (totalCUDADevice ==0){
            printf("### Disable cuda device for running in cpu. \n");
        }
    }

    timer_memcpy = timer_new("Memcpy");
    timer_commu = timer_new("Communication");
    timer_compute = timer_new("Compute");

    timer_phase3 = timer_new("Phase3");

    logFp = freopen("./log.txt","wr", stderr);

//    in1= "D:\\c\\codeblock\\c\\parallel_programming\\hw4\\hw4\\Testcase\\tiny_test_case";
//    out1="D:\\c\\codeblock\\c\\parallel_programming\\hw4\\hw4\\Testcase\\output\\tiny_test_case_out";
    //in1= "./Testcase/in2";
    //out1="./output/tiny_test_case_out";

    ///char *in1= "D:\\c\\codeblock\\c\\parallel_programming\\hw4\\hw4\\Testcase\\in3";
    ///char *out1="D:\\c\\codeblock\\c\\parallel_programming\\hw4\\hw4\\Testcase\\output\\out1";
    ///input( in1 );

    input( argv[1] );
    if(totalCUDADevice > 0) {
        hipMalloc((void**)&devDist,sizeof(int)*V*n);
    }

    /*
    fprintf(logFp, "\n");
    int i,j=0;
    for ( i = 0; i < n; ++i) {
        for ( j = 0; j < n; ++j) {
            if (Dist[i][j] >= INF)	fprintf(logFp, "INF ");
            else					fprintf(logFp, "%d ", Dist[i][j]);
        }
        fprintf(logFp, "\n");
    }
    */


    if(totalCUDADevice > 0) {
        timer_start(timer_memcpy);
        debugCUDACall(hipMemcpy(devDist,&Dist[0][0],sizeof(int)*V*n,hipMemcpyHostToDevice));
        timer_end(timer_memcpy);
    }


    int B = 128;
    if(argc > 3) {
        sscanf(argv[3],"%d",&B);
    }

    printf("*** B=%d, source=%s, output=%s,\n",B,argv[1],argv[2]);

    timer_start(timer_phase3);
    timer_end(timer_phase3);

    timer_start(timer_compute);
    timer_end(timer_compute);

    block_APSP(B);


    if(totalCUDADevice >0) {
        Timer tempMemcpy;
        timer_init(&tempMemcpy,"");
        timer_start(&tempMemcpy);
        debugCUDACall(hipMemcpy(&Dist[0][0],devDist,sizeof(int)*n*V,hipMemcpyDeviceToHost));
        timer_end(&tempMemcpy);
        timer_add(timer_memcpy,&tempMemcpy);
        debugCUDACall(hipFree(devDist));
    }
    output(argv[2]);
    ///output( out1 );
    timer_print(timer_memcpy,stdout);
    timer_print(timer_commu,stdout);
    timer_print(timer_compute,stdout);
    timer_print(timer_phase3,stdout);

    fclose(logFp);
    timer_delete(timer_memcpy);
    timer_delete(timer_commu);
    timer_delete(timer_compute);
    timer_delete(timer_phase3);
    return 0;
}

void input(char *inFileName)
{   FILE *infile = fopen(inFileName, "r");
    fscanf(infile, "%d %d", &n, &m);

    printf("n=%d, m=%d \n",n,m);
    int i,j=0;
    for ( i = 0; i < n; ++i) {
        for ( j = 0; j < n; ++j) {
            if (i == j)	Dist[i][j] = 0;
            else		Dist[i][j] = INF;
        }
    }

    while (--m >= 0) {
        int a, b, v;
        fscanf(infile, "%d %d %d", &a, &b, &v);
        if (m== 49) printf("m=%d, a=%d, b=%d, v=%d \n",m,a,b,v);
        --a, --b;
        Dist[a][b] = v;
    }
}

void output(char *outFileName)
{   FILE *outfile = fopen(outFileName, "w");
    int i,j=0;
    for ( i = 0; i < n; ++i) {
        for ( j = 0; j < n; ++j) {
            if (Dist[i][j] >= INF)	fprintf(outfile, "INF ");
            else					fprintf(outfile, "%d ", Dist[i][j]);
        }
        fprintf(outfile, "\n");
    }
}


int iceil(int a, int b) {
    return (a + b -1)/b;
}
///  (y,x) ==> (column,row)
///        |       |
///  (0,0) | (0,1) | (0,2)
///  ______|_______|______
///  (1,0) | (1,1) | (1,2)
///  ______|_______|______
///  (2,0) | (2,1) | (2,2)
///        |       |

void block_APSP(int B) {
    int round = iceil(n, B);
#if DEBUG_ENABLE
    fprintf(logFp,"round=%d ====================================\n",round);
#endif
    int r=0;
    for ( r = 0; r < round; ++r) {
        ///* Phase 1*/
#if DEBUG_ENABLE
        fprintf(logFp,"[Phase1] r=%d ====================================\n",r);
#endif
        /// B, Round, block_start_x, block_start_y, block_width, block_height
        cal(B,	r,	  r,	r,	            1,	           1);

#if DEBUG_ENABLE
        fprintf(logFp,"[Phase2] r=%d \n",r);
        fprintf(logFp,"         r=%d 1. \n",r);
#endif
        ///* Phase 2*/  /// (y,x) ==> (column,row) ==> (r,0)
        cal(B, r,     r,     0,             r,             1); ///front row
#if DEBUG_ENABLE
        fprintf(logFp,"         r=%d 2. \n",r);
#endif
        cal(B, r,     r,  r +1,  round - r -1,             1); /// back row
#if DEBUG_ENABLE
        fprintf(logFp,"         r=%d 3. \n",r);
#endif
        cal(B, r,     0,     r,             1,             r); /// up column
#if DEBUG_ENABLE
        fprintf(logFp,"         r=%d 4. \n",r);
#endif
        cal(B, r,  r +1,     r,             1,  round - r -1); /// down column
#if DEBUG_ENABLE
        fprintf(logFp,"[Phase3] r=%d \n",r);
#endif

        Timer tempTime;
        timer_init(&tempTime,"");
        timer_start(&tempTime);
#if DEBUG_ENABLE
        ///* Phase 3*/
        fprintf(logFp,"         r=%d 1. \n",r);
#endif
        cal(B, r,     0,     0,            r,             r);  ///2 quadrant
#if DEBUG_ENABLE
        fprintf(logFp,"         r=%d 2. \n",r);
#endif
        cal(B, r,     0,  r +1,  round -r -1,             r);  /// 1 quadrant
#if DEBUG_ENABLE
        fprintf(logFp,"         r=%d 3. \n",r);
#endif
        cal(B, r,  r +1,     0,            r,  round - r -1);   /// 3 quadrant
#if DEBUG_ENABLE
        fprintf(logFp,"         r=%d 4. \n",r);
#endif
        cal(B, r,  r +1,  r +1,  round -r -1,  round - r -1);   /// 4 quadrant
        timer_end(&tempTime);
        timer_add(timer_phase3,&tempTime);


        /*
        char *devOut0 ="in3_dev_0.txt";
        char *devOut1 ="in3_dev_1.txt";
        char *devOut2 ="in3_dev_0.txt";
        char *devOut3 ="in3_dev_0.txt";
        char *devOut0 ="in3_dev_0.txt";
        char *devOut0 ="in3_dev_0.txt";
        char *devOut0 ="in3_dev_0.txt";
        char *devOut0 ="in3_dev_0.txt";
        */

        #if ERROR_TRACING
        if (totalCUDADevice ==0){

            char *fileName = "in3_dev_";
            char *extFile =".txt";
            char roundString[10];

            sprintf(roundString,"%d",r);
            fileName = stringConcat(fileName, roundString);
            fileName = stringConcat(fileName, extFile);

            FILE *outfile = fopen(fileName, "w");
            fprintf(outfile, "round=%d, \n",r);

            int i,j=0;
            for ( i = 0; i < n; ++i) {
                for ( j = 0; j < n; ++j) {
                    fprintf(outfile, "[%d][%d]=%d, ",i,j,Dist[i][j]);
                }
                fprintf(outfile, "\n");
            }

            fclose(outfile);
        }
        #endif

    }
}
static __global__ void column_CalKernelGPU(int B,int Round,int x,int y,int n,int* dDist,int k) {
     ///column實際跑後，花了14sec，跑很快。
    //int Bpow2=B*B;
    int b_i = blockIdx.x+x;
    int b_j = blockIdx.y+y;
    int valIK,valKJ,valIJ;
    for(int bid=0; bid<B; bid+=1) {
        int threadIdx_x=bid;
        int threadIdx_y=threadIdx.x;
        int i=b_i*B+threadIdx_x;
        int j=b_j*B+threadIdx_y;
        if (i > n) continue;
        if (j > n) continue;
        valIK=dDist[i*V+k];
        valKJ=dDist[k*V+j];
        valIJ=dDist[i*V+j];
        if (valIK + valKJ < valIJ) {
            valIJ = valIK + valKJ;
            dDist[i*V+j]=valIJ;
        }
        //__threadfence();
    }
}


static __global__ void rwo_CalKernelGPU(int B,int Round,int x,int y,int n,int* dDist,int k) {
    ///row 實際跑後，花了294sec，跑很久。
    int b_i = blockIdx.x+x;
    int b_j = blockIdx.y+y;
    int valIK,valKJ,valIJ;
    for(int bid=0; bid<B; bid+=1) {
        int i=b_i*B+threadIdx.x;
        int j=b_j*B+bid;
        if (i > n) continue;
        if (j > n) continue;
        valIK=dDist[i*V+k];
        valKJ=dDist[k*V+j];
        valIJ=dDist[i*V+j];
        if (valIK + valKJ < valIJ) {
            valIJ = valIK + valKJ;
            dDist[i*V+j]=valIJ;
        }
        //__threadfence();
    }
}

static void calKernelCPU(int B,int Round,int b_i,int b_j) {
//////////////////////
    int k=0;
    /// To calculate B*B elements in the block (b_i, b_j)
    /// For each block, it need to compute B times
    int block_internal_start_x 	= b_i * B;
    int block_internal_end_x 	= (b_i +1) * B;
    int block_internal_start_y = b_j * B;
    int block_internal_end_y 	= (b_j +1) * B;
    if (block_internal_end_x > n)	block_internal_end_x = n;
    if (block_internal_end_y > n)	block_internal_end_y = n;
    for ( k = Round * B; k < (Round +1) * B && k < n; ++k) { ///
        int i,j;
        /// To calculate original index of elements in the block (b_i, b_j)
        /// For instance, original index of (0,0) in block (1,2) is (2,5) for V=6,B=2
        for ( i = block_internal_start_x; i < block_internal_end_x; ++i) {
            for ( j = block_internal_start_y; j < block_internal_end_y; ++j) {
                if (Dist[i][k] + Dist[k][j] < Dist[i][j])
                    Dist[i][j] = Dist[i][k] + Dist[k][j];
            }
        }
    }
}

static void calLauncherCPU(int B,int Round,int x,int y,int w,int h) {
    int b_i,b_j;
    for ( b_i =  0; b_i < h; ++b_i) {
        for ( b_j = 0; b_j < w; ++b_j) {
            calKernelCPU(B,Round,b_i+x,b_j+y);
        }
    }
}
static struct hipDeviceProp_t prop;
static int devicePropGot=0;
static void getProp() {
    if(!devicePropGot) {
        devicePropGot=1;
        hipGetDeviceProperties(&prop,currentDev);
    }
}

int isFirst = 1;
void calLauncher(int B,int Round,int x,int y,int w,int h) {
    dim3 gdim(h,w,1);
    dim3 bdim(B,1,1);
    hipError_t err;
    if(totalCUDADevice == 0) {
        if (isFirst){
            isFirst=0;
            printf("run in cpu ,because totalCUDADevice=%d\n",totalCUDADevice );
        }
        calLauncherCPU(B,Round,x,y,w,h);
        return;
    }
    int mink=Round*B;
    int maxk=mink+B;
    if(maxk>n) maxk=n;
    getProp();
    if(bdim.x > prop.maxThreadsPerBlock) {
        bdim.x=prop.maxThreadsPerBlock;
    }

    for (int k = mink; k < maxk; ++k) { ///

        Timer tempTime;
        timer_init(&tempTime,"");
        timer_start(&tempTime);
        column_CalKernelGPU<<<gdim,bdim>>>(B,Round,x,y,n,devDist,k);
        err=hipDeviceSynchronize();
        timer_end(&tempTime);
        timer_add(timer_compute,&tempTime);

        if(err != hipSuccess) {
            fprintf(stderr,"%s(gdim=%d,%d,%d)(bid=%d,%d,%d)\n",
                    hipGetErrorString(err),
                    gdim.x,gdim.y,gdim.z,bdim.x,bdim.y,bdim.z);
        }
    }
}
void cal(int B, int Round, int x,int y,int w,int h)
{
#if DEBUG_ENABLE
    int i,j=0;
    int block_end_x = x + h ;
    int block_end_y = y + w;
    fprintf(logFp,"B=%d, Round=%d, block_start_x=%d, block_start_y=%d, block_width=%d, block_height=%d, \n",B,Round,x,y,w,h);
    fprintf(logFp,"block_end_x=%d, block_end_y=%d,\n",block_end_x,block_end_y);
#endif
    calLauncher(B,Round,x,y,w,h);
#if DEBUG_ENABLE
    fprintf(logFp, "\n");
    i,j=0;
    for ( i = 0; i < n; ++i) {
        for ( j = 0; j < n; ++j) {
            if (Dist[i][j] >= INF)	fprintf(logFp, "INF ");
            else					fprintf(logFp, "%d ", Dist[i][j]);
        }
        fprintf(logFp, "\n");
    }
    fprintf(logFp, "------------------------------------------------\n");
#endif
}


int main(int argc, char* argv[]) {
    struct timeval tv, tv2;

    clock_t endTime;
    unsigned long long start_utime, end_utime;


    endTime =clock();
    gettimeofday(&tv, NULL);

    mainRun( argc, argv);

    gettimeofday(&tv2, NULL);
    endTime =clock() - endTime ;

    start_utime = tv.tv_sec * 1000000 + tv.tv_usec;
    end_utime = tv2.tv_sec * 1000000 + tv2.tv_usec;

    printf("Clock=%f sec. ,  Gettimeofday time = %llu.%03llu milisecond;  %llu.%03llu sec \n",((float)endTime) /CLOCKS_PER_SEC, (end_utime - start_utime)/1000, (end_utime - start_utime)%1000, (end_utime - start_utime)/1000000, (end_utime - start_utime)%1000000  );

    return 0;

}


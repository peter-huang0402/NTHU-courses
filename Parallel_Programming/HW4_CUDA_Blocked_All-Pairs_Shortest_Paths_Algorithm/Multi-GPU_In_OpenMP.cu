
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <sys/time.h>
#include <omp.h>
#include <sys/types.h>
#define DEBUG_ENABLE 0
#define ERROR_TRACING 0
#define V 7000
#define INF 1000000
///const int INF = 1000000;
///const int V = 7000;
void input(char *inFileName);
void output(char *outFileName);

void openMP_Block_APSP(int B);
int iceil(int a, int b);
void cal(int B, int Round, int block_start_x, int block_start_y, int block_width, int block_height , int gpuDeviceNo);

int n, m;	// Number of vertices, edges
int Dist[V][V];
int* devDist;
int* devOtherDist;
char* devOtherChange;

///char change[V][V];
char otherChange[V][V];
int otherDist[V][V];

int gpuID=0;
int  otherDeviceNo =0;

FILE *logFp;
char *in1;
char *out1;

static int totalCUDADevice = 0;
static int currentDev=0;


typedef struct Timer {
    char name[256];
    struct timeval begin;
    struct timeval end;
} Timer;

Timer* timer_memcpy,*timer_commu,*timer_compute;
Timer *timer_phase3, *timer_allPhase, *timer_main, *subcard_compute;

Timer* timer_init(Timer* t,const char* name);
Timer* timer_new(const char* name);
void timer_start(Timer* t);
void timer_end(Timer* t);
void timer_add(Timer* t1, const Timer* t2);
double timer_seconds(const Timer* t);
void timer_print(const Timer* t,FILE* stream);
void timer_delete(Timer* t);


Timer* timer_init(Timer* t,const char* name ) {
    if(t) {
        memset (t,0,sizeof(Timer));
        strncpy(t->name,name,256);
    }
    return t;
}

Timer* timer_new(const char* name) {
    Timer* t;
    t = (Timer*)malloc(sizeof(Timer));
    return timer_init(t,name);
}
void timer_start(Timer* t) {
    if(!t) return;
    gettimeofday(&t->begin,0);
}
void timer_end(Timer* t) {
    if(!t) return;
    gettimeofday(&t->end,0);
}
void timer_add(Timer* t1, const Timer* t2) {
    if(!t1 || !t2) return;
    t1->end.tv_sec+=(t2->end.tv_sec-t2->begin.tv_sec);
    t1->end.tv_usec+=(t2->end.tv_usec-t2->begin.tv_usec);
}

double timer_seconds(const Timer* t) {
    if(!t) return 0;
    return (double)(t->end.tv_sec-t->begin.tv_sec)+(1e-6*(t->end.tv_usec-t->begin.tv_usec));
}
void timer_delete(Timer* t) {
    if(!t) return;
    free(t);
}
void timer_print(const Timer* t, FILE* stream) {
    if(!t) return;
    fprintf(stream,"%s : %f(sec)\n",t->name,timer_seconds(t));
}

static void __debugCUDACall(hipError_t err, const char* expr, const char* file, int line) {
    if(err != hipSuccess) {
        fprintf(stderr,"###Error in File %s Line %d:%s\n",file,line,expr);
        fprintf(stderr,"%s \n",hipGetErrorString(err));
    }
}
#define debugCUDACall(X) __debugCUDACall((X),#X,__FILE__,__LINE__)


char *stringConcat(char *str1, char *str2) {

    int length=strlen(str1)+strlen(str2)+1;

    char *result = (char*)malloc(sizeof(char) * length);

    // 複製第一個字串至新的陣列空間
    strcpy(result, str1);
    // 串接第二個字串至新的陣列空間
    strcat(result, str2);

    return result;
}


void initCUDADevice(int gpuID)
{
    // Task 1: Device Initialization
    hipGetDeviceCount(&totalCUDADevice);
    printf("totalCUDADevice=%d, \n",totalCUDADevice);


    if (totalCUDADevice == 0) {
        printf("No CUDA device found.\n\n");
    } else if (gpuID < totalCUDADevice) {
        printf("set CUDA device=%d, \n",gpuID );
        hipSetDevice(gpuID);
    } else {
        gpuID =0;
        printf("set CUDA device=%d, \n",gpuID );
        hipSetDevice(gpuID);
    }

    if (gpuID ==0){
       otherDeviceNo =1;
    }else{
        otherDeviceNo =0;
    }
}

int mainRun(int argc, char* argv[])
{


    if(argc > 4) {
        sscanf(argv[4],"%d",&gpuID);
    }

    initCUDADevice(gpuID);



    omp_set_num_threads(totalCUDADevice);

    timer_memcpy = timer_new("Memcpy");
    timer_commu = timer_new("Communication");
    timer_compute = timer_new("Compute");
    subcard_compute = timer_new("SubCard_Compute");


    timer_phase3 = timer_new("Phase3");

    timer_allPhase= timer_new("All Phase");

    timer_main = timer_new("openMP_Block_APSP");

    logFp = freopen("./omp_log.txt","wr", stderr);

//    in1= "D:\\c\\codeblock\\c\\parallel_programming\\hw4\\hw4\\Testcase\\tiny_test_case";
//    out1="D:\\c\\codeblock\\c\\parallel_programming\\hw4\\hw4\\Testcase\\output\\tiny_test_case_out";
    //in1= "./Testcase/in2";
    //out1="./output/tiny_test_case_out";

    ///char *in1= "D:\\c\\codeblock\\c\\parallel_programming\\hw4\\hw4\\Testcase\\in3";
    ///char *out1="D:\\c\\codeblock\\c\\parallel_programming\\hw4\\hw4\\Testcase\\output\\out1";
    ///input( in1 );

    input( argv[1] );
    if(totalCUDADevice > 0) {
        hipMalloc((void**)&devDist,sizeof(Dist));

        debugCUDACall(hipSetDevice(otherDeviceNo));
        debugCUDACall(hipMalloc((void**)&devOtherChange, sizeof(otherChange)));
        debugCUDACall(hipMalloc((void**)&devOtherDist,sizeof(otherDist)));
        debugCUDACall(hipSetDevice(gpuID));
    }

    /*
    fprintf(logFp, "\n");
    int i,j=0;
    for ( i = 0; i < n; ++i) {
        for ( j = 0; j < n; ++j) {
            if (Dist[i][j] >= INF)	fprintf(logFp, "INF ");
            else					fprintf(logFp, "%d ", Dist[i][j]);
        }
        fprintf(logFp, "\n");
    }
    */


    if(totalCUDADevice > 0) {
        timer_start(timer_memcpy);
        debugCUDACall(hipMemcpy(devDist,&Dist[0][0],sizeof(int)*V*n,hipMemcpyHostToDevice));
        timer_end(timer_memcpy);
    }


    int B = 128;
    if(argc > 3) {
        sscanf(argv[3],"%d",&B);
    }

    printf("*** B=%d, source=%s, output=%s,\n",B,argv[1],argv[2]);



      timer_start(timer_allPhase);
    timer_end(timer_allPhase);

    timer_start(subcard_compute);
    timer_end(subcard_compute);

    timer_start(timer_phase3);
    timer_end(timer_phase3);

    timer_start(timer_compute);
    timer_end(timer_compute);


    timer_start(timer_main);
    openMP_Block_APSP(B);
    timer_end(timer_main);


    if(totalCUDADevice >0) {
        Timer tempMemcpy;
        timer_init(&tempMemcpy,"");
        timer_start(&tempMemcpy);
        debugCUDACall(hipMemcpy(&Dist[0][0],devDist,sizeof(int)*n*V,hipMemcpyDeviceToHost));
        timer_end(&tempMemcpy);
        timer_add(timer_memcpy,&tempMemcpy);
        debugCUDACall(hipFree(devDist));
    }

    output(argv[2]);
    ///output( out1 );
    timer_print(timer_memcpy,stdout);
    timer_print(timer_commu,stdout);
    timer_print(timer_allPhase,stdout);
    timer_print(timer_compute,stdout);


    timer_print(timer_phase3,stdout);
    timer_print(subcard_compute,stdout);
    timer_print(timer_main,stdout);

    fclose(logFp);
    timer_delete(timer_memcpy);
    timer_delete(timer_commu);
    timer_delete(timer_allPhase);
    timer_delete(timer_compute);
    timer_delete(timer_phase3);
     timer_delete(subcard_compute);
    timer_delete(timer_main);

    return 0;
}

void input(char *inFileName)
{   FILE *infile = fopen(inFileName, "r");
    fscanf(infile, "%d %d", &n, &m);

    printf("n=%d, m=%d \n",n,m);
    int i,j=0;
    for ( i = 0; i < n; ++i) {
        for ( j = 0; j < n; ++j) {
            if (i == j)	Dist[i][j] = 0;
            else		Dist[i][j] = INF;
        }
    }

    while (--m >= 0) {
        int a, b, v;
        fscanf(infile, "%d %d %d", &a, &b, &v);
        if (m== 49) printf("m=%d, a=%d, b=%d, v=%d \n",m,a,b,v);
        --a, --b;
        Dist[a][b] = v;
    }
}

void output(char *outFileName)
{   FILE *outfile = fopen(outFileName, "w");
    int i,j=0;
    for ( i = 0; i < n; ++i) {
        for ( j = 0; j < n; ++j) {
            if (Dist[i][j] >= INF)	fprintf(outfile, "INF ");
            else					fprintf(outfile, "%d ", Dist[i][j]);
        }
        fprintf(outfile, "\n");
    }
}

void readChangeAndData() {
    if(totalCUDADevice > 1) {
         # if ERROR_TRACING
        printf("readChangeAndData \n");
        fprintf(logFp, "readChangeAndData \n");
         #endif

        Timer tempMemcpy;
        timer_init(&tempMemcpy,"");
        timer_start(&tempMemcpy);

        debugCUDACall(hipMemcpy(otherDist,devOtherDist,sizeof(int)*n*V,hipMemcpyDeviceToHost));
        debugCUDACall(hipMemcpy(otherChange,devOtherChange,sizeof(char)*n*V,hipMemcpyDeviceToHost));
        timer_end(&tempMemcpy);
        timer_add(timer_memcpy,&tempMemcpy);

    }
}

void copyFormSubGPUToMainGPU(int round){
    Timer temp;
    timer_init(&temp,"");

    debugCUDACall(hipSetDevice(otherDeviceNo));
    readChangeAndData();


    Timer tempMemcpy;
    timer_init(&tempMemcpy,"");
    timer_start(&tempMemcpy);
    /// copy data form device first from main gpu
    debugCUDACall(hipSetDevice(gpuID));
    debugCUDACall(hipMemcpy(Dist,devDist,sizeof(int)*n*V,hipMemcpyDeviceToHost));

    timer_end(&tempMemcpy);
    timer_add(timer_memcpy,&tempMemcpy);


    timer_start(&temp);
    # if ERROR_TRACING
     fprintf(logFp, "round=%d, otherChange[0][2]=%d \n",round,otherChange[0][2]);
    fprintf(logFp, "\n\n####round=%d,Change Change Change Dist \n",round);
    #endif

    #pragma omp parallel for
    for(int iL=0; iL<V*n; ++iL) {
        int i = iL%V;
        int j = iL/V;
        if (i >= n) continue;
        if(otherChange[j][i]) {
               Dist[j][i]=otherDist[j][i];
               # if ERROR_TRACING
               if (round ==0) fprintf(logFp, "[%d][%d]=%d, ",j,i,Dist[j][i]);
               #endif
        }
    }


    # if ERROR_TRACING
    if (round ==1 ){
        fprintf(logFp, "\n\n####round=%d,Total Dist \n", round);
        ///#pragma omp parallel for
        for(int iL=0; iL<V*n; ++iL) {
            int i = iL%V;
            int j = iL/V;
            if (j ==0 && i== 2){
                 fprintf(logFp, "\n### [%d][%d], iL=%d, \n",j,i,iL);
            }
            if (i >= n) continue;
            fprintf(logFp, "[%d][%d]=%d, ",j,i,Dist[j][i]);
        }
    }
    #endif

    ///memset(otherChange,0,sizeof(char)*n*V);
    ///debugCUDACall(cudaMemset(devOtherChange,0,sizeof(char)*n*V));

    timer_end(&temp);
    timer_add(timer_commu,&temp);


    if(totalCUDADevice > 1) {
        Timer temp;
        timer_init(&temp,"");
        timer_start(&temp);
        debugCUDACall(hipMemcpy(devDist,Dist,sizeof(int)*n*V,hipMemcpyHostToDevice));
        timer_end(&temp);
        timer_add(timer_memcpy,&temp);
    }

}

void validateVaule(int round){


     if (totalCUDADevice > 1 ){

            debugCUDACall(hipMemcpy(Dist,devDist,sizeof(int)*n*V,hipMemcpyDeviceToHost));
            char *fileName = "omp_in3_dev_";
            char *extFile =".txt";
            char roundString[10];

            sprintf(roundString,"%d",round);
            fileName = stringConcat(fileName, roundString);
            fileName = stringConcat(fileName, extFile);

            FILE *outfile = fopen(fileName, "w");
            fprintf(outfile, "round=%d, \n",round);

            int i,j=0;
            for ( i = 0; i < n; ++i) {
                for ( j = 0; j < n; ++j) {
                    fprintf(outfile, "[%d][%d]=%d, ",i,j,Dist[i][j]);
                }
                fprintf(outfile, "\n");
            }

            fclose(outfile);
     }

}

int iceil(int a, int b) {
    return (a + b -1)/b;
}
///  (y,x) ==> (column,row)
///        |       |
///  (0,0) | (0,1) | (0,2)
///  ______|_______|______
///  (1,0) | (1,1) | (1,2)
///  ______|_______|______
///  (2,0) | (2,1) | (2,2)
///        |       |

void openMP_Block_APSP(int B) {
    int round = iceil(n, B);
#if DEBUG_ENABLE
    fprintf(logFp,"round=%d ====================================\n",round);
#endif


    int r=0;
    for ( r = 0; r < round; ++r) {

        debugCUDACall(hipSetDevice(gpuID));

         Timer phase1_2Time;
         timer_init(&phase1_2Time,"");
         timer_start(&phase1_2Time);

        ///* Phase 1*/
#if DEBUG_ENABLE
        fprintf(logFp,"[Phase1] r=%d ====================================\n",r);
#endif
        /// B, Round, block_start_x, block_start_y, block_width, block_height
        cal(B,	r,	  r,	r,	            1,	           1 , gpuID);

#if DEBUG_ENABLE
        fprintf(logFp,"[Phase2] r=%d \n",r);
        fprintf(logFp,"         r=%d 1. \n",r);
#endif
        ///* Phase 2*/  /// (y,x) ==> (column,row) ==> (r,0)
        cal(B, r,     r,     0,             r,             1 , gpuID); ///front row
#if DEBUG_ENABLE
        fprintf(logFp,"         r=%d 2. \n",r);
#endif
        cal(B, r,     r,  r +1,  round - r -1,             1 , gpuID); /// back row
#if DEBUG_ENABLE
        fprintf(logFp,"         r=%d 3. \n",r);
#endif
        cal(B, r,     0,     r,             1,             r , gpuID); /// up column
#if DEBUG_ENABLE
        fprintf(logFp,"         r=%d 4. \n",r);
#endif
        cal(B, r,  r +1,     r,             1,  round - r -1 , gpuID); /// down column
#if DEBUG_ENABLE
        fprintf(logFp,"[Phase3] r=%d \n",r);
#endif

        timer_end(&phase1_2Time);
        timer_add(timer_allPhase, &phase1_2Time);



        if(totalCUDADevice > 1){
              debugCUDACall(hipMemcpy(Dist,devDist,sizeof(int)*n*V,hipMemcpyDeviceToHost));

              # if ERROR_TRACING
              if (r ==1){
                fprintf(logFp, "\n\nround=%d, #####Finish pharse2!!\n",r);
                int i,j=0;
                for ( i = 0; i < n; ++i) {
                    for ( j = 0; j < n; ++j) {
                        fprintf(logFp, "[%d][%d]=%d, ",i,j,Dist[i][j]);
                    }
                    fprintf(logFp, "\n\n\n");
                }
              }
              # endif

              ///copy main device data to another device
              debugCUDACall(hipSetDevice(otherDeviceNo));
              memset(otherChange,0,sizeof(char)*n*V);

              Timer tempMemcpy;
              timer_init(&tempMemcpy,"");
              timer_start(&tempMemcpy);
              debugCUDACall(hipMemset(devOtherChange,0,sizeof(char)*n*V));
              debugCUDACall(hipMemcpy(devOtherDist,Dist,sizeof(int)*n*V,hipMemcpyHostToDevice));
              timer_end(&tempMemcpy);
              timer_add(timer_memcpy,&tempMemcpy);

              debugCUDACall(hipSetDevice(gpuID));
        }

        Timer tempTime;
        timer_init(&tempTime,"");
        timer_start(&tempTime);
         /// Phase 3/
         #pragma omp parallel
         {
            unsigned int cpu_thread_id = omp_get_thread_num();
            unsigned int num_cpu_threads = omp_get_num_threads();

            ///printf("Start CPU thread=%d, (total threads=%d)\n",cpu_thread_id, num_cpu_threads);

            if (cpu_thread_id == gpuID){

                 debugCUDACall(hipSetDevice(gpuID));
                 cal(B, r,     0,     0,            r,             r , gpuID);  ///2 quadrant
                 cal(B, r,     0,  r +1,  round -r -1,             r , gpuID);  /// 1 quadrant

                 if (totalCUDADevice ==1){
                       # if ERROR_TRACING
                     printf("### Error totalCUDADevice=%d \n",totalCUDADevice);
                     fprintf(logFp, "### Error totalCUDADevice=%d \n",totalCUDADevice);
                       # endif
                     cal(B, r,  r +1,     0,            r,  round - r -1 , otherDeviceNo);   /// 3 quadrant
                     cal(B, r,  r +1,  r +1,  round -r -1,  round - r -1 , otherDeviceNo);   /// 4 quadrant
                 }

            }else{
                     # if ERROR_TRACING
                 printf("Another GPU, round=%d, 3 quardrant \n",r);
                 fprintf(logFp, "Another GPU, round=%d, 3 quardrant \n",r);
                    # endif
                 debugCUDACall(hipSetDevice(otherDeviceNo));
                 cal(B, r,  r +1,     0,            r,  round - r -1 , otherDeviceNo);   /// 3 quadrant

                   # if ERROR_TRACING
                 printf("Another GPU, round=%d, 4 quardrant \n",r);
                 fprintf(logFp, "Another GPU, round=%d, 4 quardrant \n",r);
                   # endif
                 cal(B, r,  r +1,  r +1,  round -r -1,  round - r -1 , otherDeviceNo);   /// 4 quadrant
            }
             ///fprintf(logFp,"End CPU thread=%d, (total threads=%d)\n",cpu_thread_id, num_cpu_threads);
        }

        timer_end(&tempTime);
        timer_add(timer_phase3,&tempTime);
        timer_add(timer_allPhase, &tempTime);

         # if ERROR_TRACING
        fprintf(logFp,"AAAA---------------------------BBBB\n");
         # endif


        if(totalCUDADevice > 1){
          copyFormSubGPUToMainGPU(r);

          # if ERROR_TRACING
          debugCUDACall(cudaSetDevice(gpuID));
          validateVaule(r);
          # endif
        }

    }

}

static __global__ void calKernelGPU(int B,int Round,int x,int y,int n,int* dDist,int k, char* dchange) {
//////////////////////
    //int Bpow2=B*B;
    int b_i = blockIdx.x+x;
    int b_j = blockIdx.y+y;
    int valIK,valKJ,valIJ;
    for(int bid=0; bid<B; bid+=1) {
        int threadIdx_x=bid;
        int threadIdx_y=threadIdx.x;
        int i=b_i*B+threadIdx_x;
        int j=b_j*B+threadIdx_y;
        if (i > n) continue;
        if (j > n) continue;
        valIK=dDist[i*V+k];
        valKJ=dDist[k*V+j];
        valIJ=dDist[i*V+j];

        # if ERROR_TRACING
        if (Round ==1 && i==0 && j ==2 && valIJ !=1000000 ){
            printf("###[%d][%d]=valIJ=%d ",i,j, valIJ);
            if (dchange != NULL ){
                printf("dchange[%d][%d]=%d, ",i,j,dchange[i*V+j]);
            }
        # endif

        if (valIK + valKJ < valIJ) {
            valIJ = valIK + valKJ;
            dDist[i*V+j]=valIJ;

             if (dchange != NULL ){
                dchange[i*V+j]=1;  // mark a change bit
            }

            # if ERROR_TRACING
            if (dchange != NULL ){
                dchange[i*V+j]=1;  // mark a change bit
                if (Round ==1 && i==0 && j ==2) printf("sub [%d][%d]=%d, ",i,j,valIJ);
            }else{
                 if (Round ==1 && i==0 && j ==2) printf("main [%d][%d]=%d, ",i,j,valIJ);
            }
            # endif
        }
        //__threadfence();
    }
}

static void calKernelCPU(int B,int Round,int b_i,int b_j) {
//////////////////////
    int k=0;
    /// To calculate B*B elements in the block (b_i, b_j)
    /// For each block, it need to compute B times
    int block_internal_start_x 	= b_i * B;
    int block_internal_end_x 	= (b_i +1) * B;
    int block_internal_start_y = b_j * B;
    int block_internal_end_y 	= (b_j +1) * B;
    if (block_internal_end_x > n)	block_internal_end_x = n;
    if (block_internal_end_y > n)	block_internal_end_y = n;
    for ( k = Round * B; k < (Round +1) * B && k < n; ++k) { ///
        int i,j;
        /// To calculate original index of elements in the block (b_i, b_j)
        /// For instance, original index of (0,0) in block (1,2) is (2,5) for V=6,B=2
        for ( i = block_internal_start_x; i < block_internal_end_x; ++i) {
            for ( j = block_internal_start_y; j < block_internal_end_y; ++j) {
                if (Dist[i][k] + Dist[k][j] < Dist[i][j])
                    Dist[i][j] = Dist[i][k] + Dist[k][j];
            }
        }
    }
}
static void calLauncherCPU(int B,int Round,int x,int y,int w,int h) {
    int b_i,b_j;
    for ( b_i =  0; b_i < h; ++b_i) {
        for ( b_j = 0; b_j < w; ++b_j) {
            calKernelCPU(B,Round,b_i+x,b_j+y);
        }
    }
}
static struct hipDeviceProp_t prop;
static int devicePropGot=0;
static void getProp() {
    if(!devicePropGot) {
        devicePropGot=1;
        hipGetDeviceProperties(&prop,currentDev);
    }
}

void calLauncher(int B,int Round,int x,int y,int w,int h, int deviceID) {
    dim3 gdim(h,w,1);
    dim3 bdim(B,1,1);
    hipError_t err;
    if(totalCUDADevice == 0) {
        printf("run in cpu ,because totalCUDADevice=%d\n",totalCUDADevice );
        calLauncherCPU(B,Round,x,y,w,h);
        return;
    }
    int mink=Round*B;
    int maxk=mink+B;
    if(maxk>n) maxk=n;
    getProp();
    if(bdim.x > prop.maxThreadsPerBlock) {
        bdim.x=prop.maxThreadsPerBlock;
    }

    for (int k = mink; k < maxk; ++k) { ///
        if (deviceID == gpuID){
            Timer tempTime;
            timer_init(&tempTime,"");
            timer_start(&tempTime);
            calKernelGPU<<<gdim,bdim>>>(B,Round,x,y,n,devDist,k, NULL );
            err=hipDeviceSynchronize();
            timer_end(&tempTime);
            timer_add(timer_compute,&tempTime);

        }else if (deviceID == otherDeviceNo){
            Timer tempTime;
            timer_init(&tempTime,"");
            timer_start(&tempTime);

            calKernelGPU<<<gdim,bdim>>>(B,Round,x,y,n,devOtherDist,k, devOtherChange );
             err=hipDeviceSynchronize();

            timer_end(&tempTime);
            timer_add(subcard_compute,&tempTime);

        }else{
           printf("### Error Error Error ####");
        }

        err=hipDeviceSynchronize();
        if(err != hipSuccess) {
            fprintf(stderr,"%s(gdim=%d,%d,%d)(bid=%d,%d,%d)\n",
                    hipGetErrorString(err),
                    gdim.x,gdim.y,gdim.z,bdim.x,bdim.y,bdim.z);
        }
    }
}


void cal(int B, int Round, int x,int y,int w,int h, int deviceID)
{
#if DEBUG_ENABLE
    int i,j=0;
    int block_end_x = x + h ;
    int block_end_y = y + w;
    fprintf(logFp,"B=%d, Round=%d, block_start_x=%d, block_start_y=%d, block_width=%d, block_height=%d, \n",B,Round,x,y,w,h);
    fprintf(logFp,"block_end_x=%d, block_end_y=%d,\n",block_end_x,block_end_y);
#endif
    calLauncher(B,Round,x,y,w,h, deviceID);
#if DEBUG_ENABLE
    fprintf(logFp, "\n");
    i,j=0;
    for ( i = 0; i < n; ++i) {
        for ( j = 0; j < n; ++j) {
            if (Dist[i][j] >= INF)	fprintf(logFp, "INF ");
            else					fprintf(logFp, "%d ", Dist[i][j]);
        }
        fprintf(logFp, "\n");
    }
    fprintf(logFp, "------------------------------------------------\n");
#endif
}


int main(int argc, char* argv[]) {
    struct timeval tv, tv2;

    clock_t endTime;
    unsigned long long start_utime, end_utime;


    endTime =clock();
    gettimeofday(&tv, NULL);

    mainRun( argc, argv);

    gettimeofday(&tv2, NULL);
    endTime =clock() - endTime ;

    start_utime = tv.tv_sec * 1000000 + tv.tv_usec;
    end_utime = tv2.tv_sec * 1000000 + tv2.tv_usec;

    printf("Clock=%f sec. ,  Gettimeofday time = %llu.%03llu milisecond;  %llu.%03llu sec \n",((float)endTime) /CLOCKS_PER_SEC, (end_utime - start_utime)/1000, (end_utime - start_utime)%1000, (end_utime - start_utime)/1000000, (end_utime - start_utime)%1000000  );

    return 0;
}

#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <sys/time.h>
#include <sys/types.h>
#include <mpi.h>
#define DEBUG_ENABLE 0
#define V 7000
#define INF 1000000
///const int INF = 1000000;
///const int V = 7000;
#define DATARANGE (n*V)
void input(char *inFileName);
void output(char *outFileName);

void block_APSP(int B);
int iceil(int a, int b);
void cal(int B, int Round, int block_start_x, int block_start_y, int block_width, int block_height);

int n, m;	// Number of vertices, edges
int Dist[V][V];
int* devDist;
char* devChange;
char change[V][V];
char otherChange[V][V];
int otherDist[V][V];
int gpuID=0;
///int Dist[7000][7000];
FILE *logFp;
char *in1;
char *out1;
static int totalCUDADevice = 0;
static int totalNode = 1;
static int MPIID=0;
static int currentDev=0;
typedef struct Timer {
    char name[256];
    struct timeval begin;
    struct timeval end;
} Timer;
Timer* timer_memcpy,*timer_commu,*timer_compute;
Timer *timer_phase3, *timer_main;

Timer* timer_init(Timer* t,const char* name);
Timer* timer_new(const char* name);
void timer_start(Timer* t);
void timer_end(Timer* t);
void timer_add(Timer* t1, const Timer* t2);
double timer_seconds(const Timer* t);
void timer_print(const Timer* t,FILE* stream);
void timer_delete(Timer* t);


Timer* timer_init(Timer* t,const char* name ) {
    if(t) {
        memset (t,0,sizeof(Timer));
        strncpy(t->name,name,256);
    }
    return t;
}

Timer* timer_new(const char* name) {
    Timer* t;
    t = (Timer*)malloc(sizeof(Timer));
    return timer_init(t,name);
}
void timer_start(Timer* t) {
    if(!t) return;
    gettimeofday(&t->begin,0);
}
void timer_end(Timer* t) {
    if(!t) return;
    gettimeofday(&t->end,0);
}
void timer_add(Timer* t1, const Timer* t2) {
    if(!t1 || !t2) return;
    t1->end.tv_sec+=(t2->end.tv_sec-t2->begin.tv_sec);
    t1->end.tv_usec+=(t2->end.tv_usec-t2->begin.tv_usec);
}

double timer_seconds(const Timer* t) {
    if(!t) return 0;
    return (double)(t->end.tv_sec-t->begin.tv_sec)+(1e-6*(t->end.tv_usec-t->begin.tv_usec));
}
void timer_delete(Timer* t) {
    if(!t) return;
    free(t);
}
void timer_print(const Timer* t, FILE* stream) {
    if(!t) return;
    fprintf(stream,"%s : %f(sec)\n",t->name,timer_seconds(t));
    //printf("%s : %f(sec)\n",t->name,timer_seconds(t));

}

static void __debugCUDACall(hipError_t err, const char* expr, const char* file, int line) {
    if(err != hipSuccess) {
        fprintf(stdout,"in File %s Line %d:%s\n",file,line,expr);
        fprintf(stdout,"%s \n",hipGetErrorString(err));
    }
}
#define debugCUDACall(X) __debugCUDACall((X),#X,__FILE__,__LINE__)


char *stringConcat(char *str1, char *str2) {

    int length=strlen(str1)+strlen(str2)+1;

    char *result = (char*)malloc(sizeof(char) * length);

    // 複製第一個字串至新的陣列空間
    strcpy(result, str1);
    // 串接第二個字串至新的陣列空間
    strcat(result, str2);

    return result;
}


void initCUDADevice(int gpuID)
{
    // Task 1: Device Initialization
    debugCUDACall(hipGetDeviceCount(&totalCUDADevice));
    printf("totalCUDADevice=%d, \n",totalCUDADevice);


    if (totalCUDADevice == 0) {
        printf("No CUDA device found.\n\n");
    } else if (gpuID < totalCUDADevice) {
        printf("set CUDA device=%d, \n",gpuID );
        debugCUDACall(hipSetDevice(gpuID));
    } else {
        gpuID =0;
        printf("set CUDA device=%d, \n",gpuID );
        debugCUDACall(hipSetDevice(gpuID));
    }

}

int mainRun(int argc, char* argv[])
{

    if(argc > 4) {
        sscanf(argv[4],"%d",&gpuID);
    }
    MPI_Comm_size(MPI_COMM_WORLD,&totalNode);
    MPI_Comm_rank(MPI_COMM_WORLD,&MPIID);

    initCUDADevice(MPIID);

    timer_memcpy = timer_new("Memcpy");
    timer_commu = timer_new("Communication");
    timer_compute = timer_new("Compute");
    timer_phase3 = timer_new("Phase3");

    timer_main = timer_new("block_APSP");

    logFp = freopen("./mpi_log.txt","wr", stderr);

//    in1= "D:\\c\\codeblock\\c\\parallel_programming\\hw4\\hw4\\Testcase\\tiny_test_case";
//    out1="D:\\c\\codeblock\\c\\parallel_programming\\hw4\\hw4\\Testcase\\output\\tiny_test_case_out";
    //in1= "./Testcase/in2";
    //out1="./output/tiny_test_case_out";

    ///char *in1= "D:\\c\\codeblock\\c\\parallel_programming\\hw4\\hw4\\Testcase\\in3";
    ///char *out1="D:\\c\\codeblock\\c\\parallel_programming\\hw4\\hw4\\Testcase\\output\\out1";
    ///input( in1 );
    if(MPIID == 0){
         input( argv[1] );
    }

    timer_start(timer_commu);
    MPI_Bcast(&n,1,MPI_INT,0,MPI_COMM_WORLD);
    timer_end(timer_commu);

    timer_start(timer_phase3);
    timer_end(timer_phase3);
    timer_start(timer_compute);
    timer_end(timer_compute);

    printf("node %d n=%d\n",MPIID,n);

    if(totalCUDADevice > 0) {
        debugCUDACall(hipMalloc((void**)&devDist,sizeof(Dist)));
        debugCUDACall(hipMalloc((void**)&devChange, sizeof(change)));
    }

    /*
    if(MPIID == 0){
       fprintf(logFp, "\n");
       int i,j=0;
       for ( i = 0; i < n; ++i) {
           for ( j = 0; j < n; ++j) {
               if (Dist[i][j] >= INF)	fprintf(logFp, "INF ");
               else					fprintf(logFp, "%d ", Dist[i][j]);
           }
           fprintf(logFp, "\n");
       }
    }
    */

    if(totalCUDADevice > 0 && MPIID == 0) {
        timer_start(timer_memcpy);
        debugCUDACall(hipMemcpy(devDist,&Dist[0][0],sizeof(int)*DATARANGE,hipMemcpyHostToDevice));
        timer_end(timer_memcpy);
    }


    int B = 128;
    if(argc > 3) {
        sscanf(argv[3],"%d",&B);
    }

    printf("*** B=%d, source=%s, output=%s,\n",B,argv[1],argv[2]);

    timer_start(timer_main);
    block_APSP(B);
    timer_end(timer_main);


    if(totalCUDADevice >0) {
        Timer tempMemcpy;
        timer_init(&tempMemcpy,"");
        timer_start(&tempMemcpy);
        if(MPIID == 0){
           debugCUDACall(hipMemcpy(&Dist[0][0],devDist,sizeof(int)*DATARANGE,hipMemcpyDeviceToHost));
        }
        timer_end(&tempMemcpy);
        timer_add(timer_memcpy,&tempMemcpy);
        debugCUDACall(hipFree(devDist));
        debugCUDACall(hipFree(devChange));
    }

    if(MPIID==0){
          output(argv[2]);
    }
    ///output( out1 );
    if(MPIID==0)
    {
        timer_print(timer_memcpy,stdout);
        timer_print(timer_commu,stdout);
        timer_print(timer_compute,stdout);
        timer_print(timer_phase3,stdout);
        timer_print(timer_main,stdout);
        fflush(stdout);
    }
    fclose(logFp);
    timer_delete(timer_memcpy);
    timer_delete(timer_commu);
    timer_delete(timer_compute);
    timer_delete(timer_phase3);
    timer_delete(timer_main);

    return 0;
}

void input(char *inFileName)
{   FILE *infile = fopen(inFileName, "r");
    fscanf(infile, "%d %d", &n, &m);

    printf("n=%d, m=%d \n",n,m);
    int i,j=0;
    for ( i = 0; i < n; ++i) {
        for ( j = 0; j < n; ++j) {
            if (i == j)	Dist[i][j] = 0;
            else		Dist[i][j] = INF;
        }
    }

    while (--m >= 0) {
        int a, b, v;
        fscanf(infile, "%d %d %d", &a, &b, &v);
        if (m== 49) printf("m=%d, a=%d, b=%d, v=%d \n",m,a,b,v);
        --a, --b;
        Dist[a][b] = v;
    }
}

void output(char *outFileName)
{   FILE *outfile = fopen(outFileName, "w");
    int i,j=0;
    for ( i = 0; i < n; ++i) {
        for ( j = 0; j < n; ++j) {
            if (Dist[i][j] >= INF)	fprintf(outfile, "INF ");
            else					fprintf(outfile, "%d ", Dist[i][j]);
        }
        fprintf(outfile, "\n");
    }
}

int iceil(int a, int b) {
    return (a + b -1)/b;
}
///  (y,x) ==> (column,row)
///        |       |
///  (0,0) | (0,1) | (0,2)
///  ______|_______|______
///  (1,0) | (1,1) | (1,2)
///  ______|_______|______
///  (2,0) | (2,1) | (2,2)
///        |       |

void readChangeAndData() {
    if(totalCUDADevice > 0) {
        Timer tempMemcpy;
        timer_init(&tempMemcpy,"");
        timer_start(&tempMemcpy);

        debugCUDACall(hipMemcpy(Dist,devDist,sizeof(int)*DATARANGE,hipMemcpyDeviceToHost));
        debugCUDACall(hipMemcpy(change,devChange,sizeof(char)*DATARANGE,hipMemcpyDeviceToHost));
        timer_end(&tempMemcpy);
        timer_add(timer_memcpy,&tempMemcpy);

    }
}
void sendDataAndChange(int other) {
    Timer temp;
    timer_init(&temp,"");

    readChangeAndData();

    timer_start(&temp);
    MPI_Send(Dist,DATARANGE,MPI_INT,other,0x1234,MPI_COMM_WORLD);
    MPI_Send(change,DATARANGE,MPI_CHAR,other,0x1234,MPI_COMM_WORLD);
    memset(change,0,sizeof(char)*DATARANGE);
    timer_end(&temp);
    timer_add(timer_commu,&temp);
}


void applyOtherDataByChange(int other) {
    MPI_Status status;
    {
       Timer temp;
       timer_init(&temp,"");
       timer_start(&temp);
       MPI_Recv(otherDist,DATARANGE,MPI_INT,other,0x1234,MPI_COMM_WORLD,&status);
       MPI_Recv(otherChange,DATARANGE,MPI_CHAR,other,0x1234,MPI_COMM_WORLD,&status);
       if(totalCUDADevice > 0) {
           Timer temp;
           timer_init(&temp,"");
           timer_start(&temp);
           debugCUDACall(hipMemcpy(Dist,devDist,sizeof(int)*DATARANGE,hipMemcpyDeviceToHost));
           timer_end(&temp);
           timer_add(timer_memcpy,&temp);
       }

       #pragma omp parallel for
       for(int iL=0; iL<DATARANGE; ++iL) {
           int i = iL%V;
           int j = iL/V;
           if(otherChange[j][i] && Dist[j][i]!=otherDist[j][i]) {
     //          printf("Dist[%d][%d]:%d->%d\n",j,i,Dist[j][i],otherDist[j][i]);
               Dist[j][i]=otherDist[j][i];
           }
       }
       timer_end(&temp);
       timer_add(timer_commu,&temp);
    }
    if(totalCUDADevice > 0) {
        Timer temp;
        timer_init(&temp,"");
        timer_start(&temp);

        debugCUDACall(hipMemcpy(devDist,Dist,sizeof(int)*DATARANGE,hipMemcpyHostToDevice));
        debugCUDACall(hipMemset(devChange,0,sizeof(char)*DATARANGE));
        timer_end(&temp);
        timer_add(timer_memcpy,&temp);
    }
}

void block_APSP(int B) {
    int round = iceil(n, B);
#if DEBUG_ENABLE
    fprintf(logFp,"round=%d ====================================\n",round);
#endif
    int r=0;
    for ( r = 0; r < round; ++r) {
        ///* Phase 1*/
#if DEBUG_ENABLE
        fprintf(logFp,"[Phase1] r=%d ====================================\n",r);
#endif

        /// B, Round, block_start_x, block_start_y, block_width, block_height
        if(MPIID == 0){
            cal(B,	r,	  r,	r,	            1,	           1); // phase 1

#if DEBUG_ENABLE
            fprintf(logFp,"[Phase2] r=%d \n",r);
            fprintf(logFp,"         r=%d 1. \n",r);
#endif
            ///* Phase 2*/  /// (y,x) ==> (column,row) ==> (r,0)

            // vertical
            // B Round   x      y              w              h
            cal(B, r,     r,     0,             r,             1); ///front row
#if DEBUG_ENABLE
            fprintf(logFp,"         r=%d 2. \n",r);
#endif
            cal(B, r,     r,  r +1,  round - r -1,             1); /// back row
            // horizontal
#if DEBUG_ENABLE
            fprintf(logFp,"         r=%d 3. \n",r);
#endif
            cal(B, r,     0,     r,             1,             r); /// up column
#if DEBUG_ENABLE
            fprintf(logFp,"         r=%d 4. \n",r);
#endif
            cal(B, r,  r +1,     r,             1,  round - r -1); /// down column
#if DEBUG_ENABLE
            fprintf(logFp,"[Phase3] r=%d \n",r);
#endif
        }

        if(totalNode > 1){
           if(totalCUDADevice > 0 && MPIID == 0){
              Timer tempMemcpy;
              timer_init(&tempMemcpy,"");
              timer_start(&tempMemcpy);
              debugCUDACall(hipMemcpy(Dist,devDist,sizeof(int)*DATARANGE,hipMemcpyDeviceToHost));
              timer_end(&tempMemcpy);
              timer_add(timer_memcpy,&tempMemcpy);
           }

           MPI_Barrier(MPI_COMM_WORLD);

           Timer temp;
           timer_init(&temp,"");
           timer_start(&temp);
           MPI_Bcast(Dist,DATARANGE,MPI_INT,0,MPI_COMM_WORLD);
           timer_end(&temp);
           timer_add(timer_commu,&temp);

           memset(change,0,sizeof(char)*DATARANGE);
           if(totalCUDADevice > 0){
              if(MPIID==1){
                 Timer tempMemcpy;
                 timer_init(&tempMemcpy,"");
                 timer_start(&tempMemcpy);
                 debugCUDACall(hipMemcpy(devDist,Dist,sizeof(int)*DATARANGE,hipMemcpyHostToDevice));
                 debugCUDACall(hipMemset(devChange,0,sizeof(char)*DATARANGE));
                 timer_end(&tempMemcpy);
                 timer_add(timer_memcpy,&tempMemcpy);
              }
           }
        }
        // here broadcast

        Timer tempTime;
        timer_init(&tempTime,"");
        timer_start(&tempTime);

        if(MPIID == 0){
#if DEBUG_ENABLE
            ///* Phase 3*/
            fprintf(logFp,"         r=%d 1. \n",r);
#endif
            cal(B, r,     0,     0,            r,             r);  ///2 quadrant
#if DEBUG_ENABLE
            fprintf(logFp,"         r=%d 2. \n",r);
#endif
            cal(B, r,     0,  r +1,  round -r -1,             r);  /// 1 quadrant
#if DEBUG_ENABLE
            fprintf(logFp,"         r=%d 3. \n",r);
#endif
            if(totalNode < 2){
               cal(B, r,  r +1,     0,            r,  round - r -1);   /// 3 quadrant
               cal(B, r,  r +1,  r +1,  round -r -1,  round - r -1);   /// 4 quadrant
            }

        }else{
           //memset(change,0,sizeof(char)*DATARANGE);
           cal(B, r,  r +1,     0,            r,  round - r -1);   /// 3 quadrant
           cal(B, r,  r +1,  r +1,  round -r -1,  round - r -1);   /// 4 quadrant

#if DEBUG_ENABLE
            fprintf(logFp,"         r=%d 4. \n",r);
#endif
        }

        timer_end(&tempTime);
        timer_add(timer_phase3,&tempTime);

        //MPI_Barrier(MPI_COMM_WORLD);
        if(totalNode > 1){
           if(MPIID==0){
              applyOtherDataByChange(1);
           }
           else if(MPIID==1){
              sendDataAndChange(0);
           }
           //MPI_Bcast(Dist,DATARANGE,MPI_INT,0,MPI_COMM_WORLD);
           //MPI_Barrier(MPI_COMM_WORLD);
        }
    }
}
static __global__ void calKernelGPU(int B,int Round,int x,int y,int n,int* dDist,int k,char* dchange) {
//////////////////////
    //int Bpow2=B*B;
    int b_i = blockIdx.x+x;
    int b_j = blockIdx.y+y;
    int valIK,valKJ,valIJ;
    for(int bid=0; bid<B; bid+=1) {
        int threadIdx_x=bid;
        int threadIdx_y=threadIdx.x;
        int i=b_i*B+threadIdx_x;
        int j=b_j*B+threadIdx_y;
        if (i > n) continue;
        if (j > n) continue;
        valIK=dDist[i*V+k];
        valKJ=dDist[k*V+j];
        valIJ=dDist[i*V+j];
        if (valIK + valKJ < valIJ) {
            valIJ = valIK + valKJ;
            dDist[i*V+j]=valIJ;
            dchange[i*V+j]=1;  // mark a change bit
        }
        //__threadfence();
    }
}

static void calKernelCPU(int B,int Round,int b_i,int b_j) {
//////////////////////
    int k=0;
    /// To calculate B*B elements in the block (b_i, b_j)
    /// For each block, it need to compute B times
    int block_internal_start_x 	= b_i * B;
    int block_internal_end_x 	= (b_i +1) * B;
    int block_internal_start_y = b_j * B;
    int block_internal_end_y 	= (b_j +1) * B;
    if (block_internal_end_x > n)	block_internal_end_x = n;
    if (block_internal_end_y > n)	block_internal_end_y = n;
    for ( k = Round * B; k < (Round +1) * B && k < n; ++k) { ///
        int i,j;
        /// To calculate original index of elements in the block (b_i, b_j)
        /// For instance, original index of (0,0) in block (1,2) is (2,5) for V=6,B=2
        for ( i = block_internal_start_x; i < block_internal_end_x; ++i) {
            for ( j = block_internal_start_y; j < block_internal_end_y; ++j) {
                if (Dist[i][k] + Dist[k][j] < Dist[i][j]) {
                    Dist[i][j] = Dist[i][k] + Dist[k][j];
                    change[i][j]=1; // mark a change bit
                }
            }
        }
    }
}
static void calLauncherCPU(int B,int Round,int x,int y,int w,int h) {
    int b_i,b_j;
    for ( b_i =  0; b_i < h; ++b_i) {
        for ( b_j = 0; b_j < w; ++b_j) {
            calKernelCPU(B,Round,b_i+x,b_j+y);
        }
    }

}
static struct hipDeviceProp_t prop;
static int devicePropGot=0;
static void getProp() {
    if(!devicePropGot) {
        devicePropGot=1;
        hipGetDeviceProperties(&prop,currentDev);
    }
}

void calLauncher(int B,int Round,int x,int y,int w,int h) {
    dim3 gdim(h,w,1);
    dim3 bdim(B,1,1);
    hipError_t err;
    if(totalCUDADevice == 0) {
        printf("run in cpu ,because totalCUDADevice=%d\n",totalCUDADevice );
        calLauncherCPU(B,Round,x,y,w,h);
        return;
    }
    int mink=Round*B;
    int maxk=mink+B;
    if(maxk>n) maxk=n;
    getProp();
    if(bdim.x > prop.maxThreadsPerBlock) {
        bdim.x=prop.maxThreadsPerBlock;
    }

    for (int k = mink; k < maxk; ++k) { ///

        Timer tempTime;
        timer_init(&tempTime,"");
        timer_start(&tempTime);
        calKernelGPU<<<gdim,bdim>>>(B,Round,x,y,n,devDist,k,devChange);
        err=hipDeviceSynchronize();
        timer_end(&tempTime);
        timer_add(timer_compute,&tempTime);

        if(err != hipSuccess) {
            fprintf(stderr,"%s(gdim=%d,%d,%d)(bid=%d,%d,%d)\n",
                    hipGetErrorString(err),
                    gdim.x,gdim.y,gdim.z,bdim.x,bdim.y,bdim.z);
        }
    }
}
void cal(int B, int Round, int x,int y,int w,int h)
{
#if DEBUG_ENABLE
    int i,j=0;
    int block_end_x = x + h ;
    int block_end_y = y + w;
    fprintf(logFp,"B=%d, Round=%d, block_start_x=%d, block_start_y=%d, block_width=%d, block_height=%d, \n",B,Round,x,y,w,h);
    fprintf(logFp,"block_end_x=%d, block_end_y=%d,\n",block_end_x,block_end_y);
#endif
    calLauncher(B,Round,x,y,w,h);
#if DEBUG_ENABLE
    fprintf(logFp, "\n");
    i,j=0;
    for ( i = 0; i < n; ++i) {
        for ( j = 0; j < n; ++j) {
            if (Dist[i][j] >= INF)	fprintf(logFp, "INF ");
            else					fprintf(logFp, "%d ", Dist[i][j]);
        }
        fprintf(logFp, "\n");
    }
    fprintf(logFp, "------------------------------------------------\n");
#endif
}


int main(int argc, char* argv[]) {
    struct timeval tv, tv2;

    clock_t endTime;
    unsigned long long start_utime, end_utime;
    MPI_Init(&argc, &argv);


    endTime =clock();
    gettimeofday(&tv, NULL);

    mainRun( argc, argv);

    gettimeofday(&tv2, NULL);
    endTime =clock() - endTime ;

    start_utime = tv.tv_sec * 1000000 + tv.tv_usec;
    end_utime = tv2.tv_sec * 1000000 + tv2.tv_usec;

    if(MPIID==0){
        printf("Clock=%f sec. ,  Gettimeofday time = %llu.%03llu milisecond;  %llu.%03llu sec \n",((float)endTime) /CLOCKS_PER_SEC, (end_utime - start_utime)/1000, (end_utime - start_utime)%1000, (end_utime - start_utime)/1000000, (end_utime - start_utime)%1000000  );
    }

    MPI_Finalize();

    return 0;

}

